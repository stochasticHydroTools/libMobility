/*Raul P. Pelaez 2021-2025. Example usage of the NBody mobility solver.
 All available solvers are used in a similar way, providing, in each case, the required parameters.
 For instance, a triply periodic algorithm will need at least a box size.
 */
#include"MobilityInterface/MobilityInterface.h"
#include"solvers/NBody/mobility.h"
#include"solvers/PSE/mobility.h"
#include <type_traits>
#include<vector>
#include<random>
#include<algorithm>
#include<iostream>
using namespace std;

using scalar = libmobility::real;
using MobilityBase = libmobility::Mobility;
using Configuration = libmobility::Configuration;
using libmobility::Parameters;
//Configures, initializes any solver (between PSE and NBody)
//The same function can be extended to create any solver.
//We need it to desambiguate by calling the solver-dependent setParameters function when necessary. For instance, see PSE below
template<class Solver>
auto initializeSolver(Parameters par){
  std::shared_ptr<MobilityBase> solver;
  if(std::is_same<Solver,NBody>::value){
    auto nbody = std::make_shared<NBody>(Configuration{.periodicityX = libmobility::periodicity_mode::open,
						   .periodicityY = libmobility::periodicity_mode::open,
						   .periodicityZ = libmobility::periodicity_mode::open});
    nbody->setParametersNBody({nbody_rpy::algorithm::advise, 1,par.numberParticles});
    solver = nbody;
  }
  if(std::is_same<Solver,PSE>::value){
    auto pse = std::make_shared<PSE>(Configuration{.periodicityX = libmobility::periodicity_mode::periodic,
						   .periodicityY = libmobility::periodicity_mode::periodic,
						   .periodicityZ = libmobility::periodicity_mode::periodic});
    scalar lx,ly,lz;
    lx=ly=lz=128;
    scalar split = 1.0;
    scalar shearStrain = 0.0;
    pse->setParametersPSE({split, lx,ly,lz, shearStrain});
    solver = pse;
  }
  solver->initialize(par);
  return solver;
}

//An example of a function that works for any solver
auto computeMFWithSolver(std::shared_ptr<MobilityBase> solver,
			 std::vector<scalar> &ipos,
			 std::vector<scalar> &iforces){
  std::vector<scalar> result(ipos.size(), 0);
  solver->setPositions(ipos);
  solver->Mdot(iforces, {}, result, {});
  return result;
}

//Lets compute the deterministic and stochastic displacements of a group of particles
int main(){

  //Create some arbitrary positions and forces
  int numberParticles = 10;
  std::vector<scalar> pos(3*numberParticles);
  auto forces = pos;
  mt19937 mersenne_engine {1234};
  uniform_real_distribution<scalar> dist {-10, 10};
  std::generate(pos.begin(), pos.end(),[&](){return dist(mersenne_engine);});
  std::generate(forces.begin(), forces.end(),[&](){return dist(mersenne_engine);});

  //Set up parameters generic to any solver
  Parameters par;
  par.hydrodynamicRadius = {1};
  par.viscosity = 1;
  par.numberParticles = numberParticles;
  par.tolerance = 1e-4;
  par.temperature = 1.0;
  par.needsTorque = false;

  //Create two different solvers
  auto solver_pse = initializeSolver<PSE>(par);
  auto solver_nbody = initializeSolver<NBody>(par);

  //Compute the displacements
  auto resultNBody = computeMFWithSolver(solver_nbody, pos, forces);
  auto resultPSE = computeMFWithSolver(solver_pse, pos, forces);

  //The solvers can be used to compute stochastic displacements, even if they do not provide a specific way to compute them (defaults to using the lanczos algorithm
  std::vector<scalar> noiseNBody(pos.size(), 0);
  scalar prefactor = 1.0;
  solver_nbody->sqrtMdotW(noiseNBody, {}, prefactor);

  //Remember to clean up when done
  solver_nbody->clean();
  solver_pse->clean();

  std::cout<<"NBody\tPSE\n";
  for(int i = 0;  i<resultNBody.size(); i++){
    std::cout<<resultNBody[i]<<"\t"<<resultPSE[i]<<std::endl;
  }
  return 0;
}
