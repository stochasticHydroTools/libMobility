/*Raul P. Pelaez 2022. Python wrapper for the DPStokes module
 */
#include "mobility.h"
#include <MobilityInterface/pythonify.h>
using DPStokesParameters = uammd_dpstokes::PyParameters;
static const char *setparameters_docstring = R"pbdoc(

When the periodicity is set to :code:`single_wall` a wall in the bottom of the domain is added.
When the periodicity is set to :code:`two_walls` a wall in the bottom and top of the domain is added.

Even in open mode (Z periodicity set to `open`) the values of :code:`zmin` and :code:`zmax` are still required. The algorithm needs to define a grid in the z direction, and these values define the extents of that grid. The code will fail if a position outside of these extents is used.

Parameters
----------
Lx : float
		The box size in the x direction.
Ly : float
		The box size in the y direction.
zmin : float
		The minimum value of the z coordinate. This is the position of the bottom wall if the Z periodicity is `single_wall` or `two_walls`.
zmax : float
		The maximum value of the z coordinate. This is the position of the top wall if the Z periodicity is `two_walls`.
allowChangingBoxSize : bool
    Whether the periodic extents Lx & Ly can be modified during parameter selection. Default: false.
)pbdoc";

static const char *docstring = R"pbdoc(
In the Doubly periodic Stokes geometry (DPStokes), an incompressible fluid exists in a domain which is periodic in the plane and open (or walled) in the third direction. The algorithm is described in [1].

The periodicity must be set to `periodic` in the X and Y directions. The Z periodicity can be set to `open`, `single_wall`, or `two_walls`. The `open` option allows for an open boundary condition in the Z direction, while `single_wall` and `two_walls` add walls at the bottom and/or top of the simulation box.

**References**

[1] Aref Hashemi, Raúl P. Peláez, Sachin Natesh, Brennan Sprinkle, Ondrej Maxian, Zecheng Gan, Aleksandar Donev; Computing hydrodynamic interactions in confined doubly periodic geometries in linear time. J. Chem. Phys. 21 April 2023; 158 (15): 154101. https://doi.org/10.1063/5.0141371
)pbdoc";

MOBILITY_PYTHONIFY_WITH_EXTRA_CODE(DPStokes,
                                   solver.def(
                                       "setParameters",
                                       [](DPStokes &self, real Lx,
                                          real Ly, real zmin, real zmax, bool allowChangingBoxSize)
                                       {
                                           DPStokesParameters params;
                                           params.Lx = Lx;
                                           params.Ly = Ly;
                                           params.zmin = zmin;
                                           params.zmax = zmax;
                                           params.allowChangingBoxSize = allowChangingBoxSize;
                                           self.setParametersDPStokes(params);
                                       },
                                       "Lx"_a, "Ly"_a, "zmin"_a,
					      "zmax"_a, "allowChangingBoxSize"_a = false, setparameters_docstring);
                                   , docstring);
